#include "hip/hip_runtime.h"
#include <ATen/ATen.h>
#include <ATen/TensorUtils.h>
#include <ATen/cuda/HIPContext.h>
#include <c10/cuda/CUDAGuard.h>
#include <ATen/cuda/CUDAApplyUtils.cuh>

#include "cuda_helpers.h"

template <typename T>
__global__ void RoIPoolForward(
    const int nthreads,
    const T* input,
    const T spatial_scale,
    const int channels,
    const int height,
    const int width,
    const int pooled_height,
    const int pooled_width,
    const T* rois,
    T* output,
    int* argmax_data) {
  CUDA_1D_KERNEL_LOOP(index, nthreads) {
    // (n, c, ph, pw) is an element in the pooled output
    int pw = index % pooled_width;
    int ph = (index / pooled_width) % pooled_height;
    int c = (index / pooled_width / pooled_height) % channels;
    int n = index / pooled_width / pooled_height / channels;

    const T* offset_rois = rois + n * 5;
    int roi_batch_ind = offset_rois[0];
    int roi_start_w = round(offset_rois[1] * spatial_scale);
    int roi_start_h = round(offset_rois[2] * spatial_scale);
    int roi_end_w = round(offset_rois[3] * spatial_scale);
    int roi_end_h = round(offset_rois[4] * spatial_scale);

    // Force malformed ROIs to be 1x1
    int roi_width = max(roi_end_w - roi_start_w + 1, 1);
    int roi_height = max(roi_end_h - roi_start_h + 1, 1);
    T bin_size_h = static_cast<T>(roi_height) / static_cast<T>(pooled_height);
    T bin_size_w = static_cast<T>(roi_width) / static_cast<T>(pooled_width);

    int hstart = static_cast<int>(floor(static_cast<T>(ph) * bin_size_h));
    int wstart = static_cast<int>(floor(static_cast<T>(pw) * bin_size_w));
    int hend = static_cast<int>(ceil(static_cast<T>(ph + 1) * bin_size_h));
    int wend = static_cast<int>(ceil(static_cast<T>(pw + 1) * bin_size_w));

    // Add roi offsets and clip to input boundaries
    hstart = min(max(hstart + roi_start_h, 0), height);
    hend = min(max(hend + roi_start_h, 0), height);
    wstart = min(max(wstart + roi_start_w, 0), width);
    wend = min(max(wend + roi_start_w, 0), width);
    bool is_empty = (hend <= hstart) || (wend <= wstart);

    // Define an empty pooling region to be zero
    T maxval = is_empty ? 0 : -FLT_MAX;
    // If nothing is pooled, argmax = -1 causes nothing to be backprop'd
    int maxidx = -1;
    const T* offset_input =
        input + (roi_batch_ind * channels + c) * height * width;
    for (int h = hstart; h < hend; ++h) {
      for (int w = wstart; w < wend; ++w) {
        int input_index = h * width + w;
        if (offset_input[input_index] > maxval) {
          maxval = offset_input[input_index];
          maxidx = input_index;
        }
      }
    }
    output[index] = maxval;
    argmax_data[index] = maxidx;
  }
}

template <typename T>
__global__ void RoIPoolBackward(
    const int nthreads,
    const T* grad_output,
    const int* argmax_data,
    const int num_rois,
    const T spatial_scale,
    const int channels,
    const int height,
    const int width,
    const int pooled_height,
    const int pooled_width,
    T* grad_input,
    const T* rois,
    const int n_stride,
    const int c_stride,
    const int h_stride,
    const int w_stride) {
  CUDA_1D_KERNEL_LOOP(index, nthreads) {
    // (n, c, ph, pw) is an element in the pooled output
    int pw = index % pooled_width;
    int ph = (index / pooled_width) % pooled_height;
    int c = (index / pooled_width / pooled_height) % channels;
    int n = index / pooled_width / pooled_height / channels;

    const T* offset_rois = rois + n * 5;
    int roi_batch_ind = offset_rois[0];
    T* grad_input_offset =
        grad_input + ((roi_batch_ind * channels + c) * height * width);

    int output_offset = n * n_stride + c * c_stride;
    const int* argmax_data_offset =
        argmax_data + (n * channels + c) * pooled_height * pooled_width;
    int argmax = argmax_data_offset[ph * pooled_width + pw];

    if (argmax != -1) {
      atomicAdd(
          grad_input_offset + argmax,
          static_cast<T>(
              grad_output[output_offset + ph * h_stride + pw * w_stride]));
    }
  }
}

std::tuple<at::Tensor, at::Tensor> ROIPool_forward_cuda(
    const at::Tensor& input,
    const at::Tensor& rois,
    const float spatial_scale,
    const int pooled_height,
    const int pooled_width) {
  AT_ASSERTM(input.device().is_cuda(), "input must be a CUDA tensor");
  AT_ASSERTM(rois.device().is_cuda(), "rois must be a CUDA tensor");

  at::TensorArg input_t{input, "input", 1}, rois_t{rois, "rois", 2};

  at::CheckedFrom c = "ROIPool_forward_cuda";
  at::checkAllSameGPU(c, {input_t, rois_t});
  at::checkAllSameType(c, {input_t, rois_t});

  at::cuda::CUDAGuard device_guard(input.device());

  auto num_rois = rois.size(0);
  auto channels = input.size(1);
  auto height = input.size(2);
  auto width = input.size(3);

  at::Tensor output = at::zeros(
      {num_rois, channels, pooled_height, pooled_width}, input.options());
  at::Tensor argmax = at::zeros(
      {num_rois, channels, pooled_height, pooled_width},
      input.options().dtype(at::kInt));

  auto output_size = num_rois * pooled_height * pooled_width * channels;
  hipStream_t stream = at::cuda::getCurrentCUDAStream();

  dim3 grid(std::min(
      at::cuda::ATenCeilDiv(
          static_cast<int64_t>(output_size), static_cast<int64_t>(512)),
      static_cast<int64_t>(4096)));
  dim3 block(512);

  if (output.numel() == 0) {
    AT_CUDA_CHECK(hipGetLastError());
    return std::make_tuple(output, argmax);
  }

  AT_DISPATCH_FLOATING_TYPES_AND_HALF(input.type(), "ROIPool_forward", [&] {
    RoIPoolForward<scalar_t><<<grid, block, 0, stream>>>(
        output_size,
        input.contiguous().data<scalar_t>(),
        spatial_scale,
        channels,
        height,
        width,
        pooled_height,
        pooled_width,
        rois.contiguous().data<scalar_t>(),
        output.data<scalar_t>(),
        argmax.data<int>());
  });
  AT_CUDA_CHECK(hipGetLastError());
  return std::make_tuple(output, argmax);
}

at::Tensor ROIPool_backward_cuda(
    const at::Tensor& grad,
    const at::Tensor& rois,
    const at::Tensor& argmax,
    const float spatial_scale,
    const int pooled_height,
    const int pooled_width,
    const int batch_size,
    const int channels,
    const int height,
    const int width) {
  // Check if input tensors are CUDA tensors
  AT_ASSERTM(grad.device().is_cuda(), "grad must be a CUDA tensor");
  AT_ASSERTM(rois.device().is_cuda(), "rois must be a CUDA tensor");
  AT_ASSERTM(argmax.device().is_cuda(), "argmax must be a CUDA tensor");

  at::TensorArg grad_t{grad, "grad", 1}, rois_t{rois, "rois", 2},
      argmax_t{argmax, "argmax", 3};

  at::CheckedFrom c = "ROIPool_backward_cuda";
  at::checkAllSameGPU(c, {grad_t, rois_t, argmax_t});
  at::checkAllSameType(c, {grad_t, rois_t});

  at::cuda::CUDAGuard device_guard(grad.device());

  auto num_rois = rois.size(0);

  at::Tensor grad_input =
      at::zeros({batch_size, channels, height, width}, grad.options());

  hipStream_t stream = at::cuda::getCurrentCUDAStream();

  dim3 grid(std::min(
      at::cuda::ATenCeilDiv(
          static_cast<int64_t>(grad.numel()), static_cast<int64_t>(512)),
      static_cast<int64_t>(4096)));
  dim3 block(512);

  // handle possibly empty gradients
  if (grad.numel() == 0) {
    AT_CUDA_CHECK(hipGetLastError());
    return grad_input;
  }

  int n_stride = grad.stride(0);
  int c_stride = grad.stride(1);
  int h_stride = grad.stride(2);
  int w_stride = grad.stride(3);

  AT_DISPATCH_FLOATING_TYPES_AND_HALF(grad.type(), "ROIPool_backward", [&] {
    RoIPoolBackward<scalar_t><<<grid, block, 0, stream>>>(
        grad.numel(),
        grad.data<scalar_t>(),
        argmax.contiguous().data<int>(),
        num_rois,
        spatial_scale,
        channels,
        height,
        width,
        pooled_height,
        pooled_width,
        grad_input.data<scalar_t>(),
        rois.contiguous().data<scalar_t>(),
        n_stride,
        c_stride,
        h_stride,
        w_stride);
  });
  AT_CUDA_CHECK(hipGetLastError());
  return grad_input;
}
